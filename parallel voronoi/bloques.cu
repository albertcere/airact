#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>


#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

typedef struct _seed
{
    unsigned char color[3];
    int x;
    int y;
} seed;

__global__ void voronoi_cuda(int count, seed* seeds, int width, int height, unsigned char* image)
{ 
    //Index 
    int x = (blockIdx.x*blockDim.x) + threadIdx.x;
    int y = (blockIdx.y*blockDim.y) + threadIdx.y;
    __shared__ seed shared_seeds[32];
    unsigned int distance;
    unsigned char tempColor[3];
    int xdistance, ydistance;
    int index = y*width*3+x*3; //Index on the MAT of RGBs
    unsigned int currentdistance = 0xFFFFFFFF;
    
    for(int i = 0; i < count; i+=32)
    {
        //Share 32 seeds with the block
        if(threadIdx.y == 0) shared_seeds[threadIdx.x] = seeds[i+threadIdx.x];

        __syncthreads();
    	#pragma unroll 32
    	for(int j = 0; j < 32; j++)
        {
    	    xdistance = shared_seeds[j].x -x;
    	    ydistance = shared_seeds[j].y -y;
    	    distance = (xdistance*xdistance)+(ydistance*ydistance);
            //If new distance < current distance, update colour
    	    if(distance < currentdistance)
            {
                currentdistance = distance;
                tempColor[0] = shared_seeds[j].color[0];
                tempColor[1] = shared_seeds[j].color[1];
                tempColor[2] = shared_seeds[j].color[2];
    	    }
            __syncthreads();
    	}
    }
    //Update global colour
    image[index] = tempColor[0];
    image[index+1] = tempColor[1];
    image[index+2] = tempColor[2];
}




static void Usage()
{
    printf("Usage: main [options]\n");
    printf("\t-n <num points>\n");
    printf("\t-i <inputfile>\t\tA list of 2-tuples (float, float) representing 2-d coordinates\n");
    printf("\t-o <outputfile.png>\n");
    printf("\t-w <width>\n");
    printf("\t-h <height>\n");
    printf("\t-p <pinned>\n");
}

int main(int argc, const char** argv) 
{
  //CUDA DECLARACIONES  
  unsigned int nBlocks, nThreads;
  
  //para calcular el tiempo en cuda
  float TiempoTotal, TiempoKernel;
  
  hipEvent_t E0, E1, E2, E3;

  
    hipEventCreate(&E0);
    hipEventCreate(&E1);
    hipEventCreate(&E2);
    hipEventCreate(&E3);
  

    //Default values
    
    //Number of seeds
    int count = 64;
    
    // Image dimension
    int width = 512;
    int height = 512;
    int pinned = 1;
    
    //Files
    const char* inputfile = 0;
    const char* outputfile = "bloques.png";
    
    /*Change default values
        -i inputfyle
        -o outputfyle
        -n count
        -w width
        -h height
        -p pinned
    */  
    for( int i = 1; i < argc; ++i )
    {
        if(strcmp(argv[i], "-i") == 0)
        {
            if( i+1 < argc ) inputfile = argv[i+1];
            else
            {
                Usage();
                return 1;
            }
        }
        else if(strcmp(argv[i], "-o") == 0)
        {
            if( i+1 < argc ) outputfile = argv[i+1];
            else
            {
                Usage();
                return 1;
            }
        }
        else if(strcmp(argv[i], "-n") == 0)
        {
            if( i+1 < argc ) count = (int)atol(argv[i+1]);
            else
            {
                Usage();
                return 1;
            }
        }
        else if(strcmp(argv[i], "-w") == 0)
        {
            if( i+1 < argc ) width = (int)atol(argv[i+1]);
            else
            {
                Usage();
                return 1;
            }
        }
        else if(strcmp(argv[i], "-h") == 0)
        {
            if( i+1 < argc ) height = (int)atol(argv[i+1]);
            else
            {
                Usage();
                return 1;
            }
        }
        else if(strcmp(argv[i], "-p") == 0)
        {
            if( i+1 < argc ) pinned = (int)atol(argv[i+1]); // si es 1 es pinned, si es 0 no.
            else
            {
                Usage();
                return 1;
            }
        }
    }



    //Open file
    FILE* file = 0;
    if( inputfile )
    {
        if( strcmp(inputfile, "-") == 0 ) file = stdin;
        file = fopen(inputfile, "r");
        if( !file )
        {
            fprintf(stderr, "Failed to open %s for reading\n", inputfile);
            return 1;
        }
        if(fscanf(file, "%d", &count)) printf("Points = %d\n", count);
        else printf("Failed to read number of seeds\n");
    }

    printf("BLOQUES with %d seeds, %d pixels of width and %d pixels of height \n", count, width, height);

    seed* seeds;
    size_t seedssize = (size_t)(sizeof(seed) * (size_t)count);
    if(pinned) hipHostMalloc((seed**)&seeds, seedssize);
    else seeds = (seed*)malloc(seedssize);
  
    
    //Get points from file
    if(inputfile)
    {
        for (int i = 0; i < count; ++i) 
        {
            if(fscanf(file,"%d %d", &seeds[i].x, &seeds[i].y));
        }
    }
    //If no input file, generate points
    else
    {
        int pointoffset = 10; // move the points inwards, for aestetic reasons

        srand(0);

        for( int i = 0; i < count; ++i )
        {
            seeds[i].x = (float)(pointoffset + rand() % (width-2*pointoffset));
            seeds[i].y = (float)(pointoffset + rand() % (height-2*pointoffset));
        }
    }


    //Generate colors for each seed
    for(int i = 0; i < count; ++i)
    {
        unsigned char basecolor = 120;
        seeds[i].color[0] = basecolor + (unsigned char)(rand() % (235 - basecolor));
        seeds[i].color[1] = basecolor + (unsigned char)(rand() % (235 - basecolor));
        seeds[i].color[2] = basecolor + (unsigned char)(rand() % (235 - basecolor));
    }



    //Initialize image
    
    size_t imagesize = (size_t)(width*height*3); //3 is for the RGB color of each pixel
    unsigned char* image;
    if(pinned) hipHostMalloc((unsigned char**)&image,imagesize);
    else image = (unsigned char*)malloc(imagesize);
    memset(image, 0, imagesize);

    hipEventRecord(E0, 0);
    hipEventSynchronize(E0);
    
    seed* device_seeds;
    hipMalloc((seed**)&device_seeds, seedssize); 

    unsigned char* device_image;
    hipMalloc((char**)&device_image, imagesize); 
  
    hipMemcpy(device_seeds, seeds, seedssize, hipMemcpyHostToDevice); //Necesario?
    hipMemcpy(device_image, image, imagesize, hipMemcpyHostToDevice);


    nThreads = 32; 
    nBlocks = width/nThreads; 

    dim3 dimGrid(nBlocks, nBlocks, 1);
    dim3 dimBlock(nThreads, nThreads, 1); 


    hipEventRecord(E1, 0);
    hipEventSynchronize(E1);

    //Fill the image
    voronoi_cuda<<<dimGrid, dimBlock>>>(count, device_seeds, width, height, device_image);

    hipEventRecord(E2, 0);
    hipEventSynchronize(E2);

    hipMemcpy(image, device_image, imagesize, hipMemcpyDeviceToHost); 

    hipFree(device_seeds);
    hipFree(device_image);
    
    hipEventRecord(E3, 0);
    hipEventSynchronize(E3);
    
    hipEventElapsedTime(&TiempoTotal,  E0, E3);
    hipEventElapsedTime(&TiempoKernel, E1, E2);
    
    printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
    printf("Tiempo Kernel: %4.6f milseg\n", TiempoKernel);
    hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);

    //Paint the seeds
    unsigned char color_seed[] = {255, 255, 255};
    int index;
    for(int i = 0; i < count; i++)
    {
        index = seeds[i].y*width*3+seeds[i].x*3;
        image[index] = color_seed[0];
        image[index+1] = color_seed[1];
        image[index+2] = color_seed[2];
    }


    //Transform tu png
    char path[512];
    sprintf(path, "%s", outputfile);
    stbi_write_png(path, width, height, 3, image, width*3);
    printf("Wrote image in %s\n", path);

    if (pinned) {
        hipHostFree(image); hipHostFree(seeds);
    }
    else {
        free(image); free(seeds);
    }
        
}
