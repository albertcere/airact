#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>


//#include <resource.h> // hay qe poner el include

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

typedef struct _seed
{
    unsigned char color[3];
    int x;
    int y;
} seed;

__global__ void voronoi_cuda(int count, seed* seeds, int width, int height, unsigned char* image, int d)
{ 
    //Index
    int x = (blockIdx.x*blockDim.x) + threadIdx.x;
    int y = (blockIdx.y*blockDim.y) + threadIdx.y;

    //Global index
    int xglobal = x;
    int yglobal = y;
    if(d == 1) yglobal = y + height;
    if(d == 2) yglobal = y + 2*height;
    if(d == 3) yglobal = y + 3*height;

    __shared__ seed shared_seeds[32];
    unsigned int distance;
    unsigned char tempColor[3];
    int xdistance, ydistance;
    int index = y*width*3+x*3; //Index on the MAT of RGBs
    unsigned int currentdistance = 0xFFFFFFFF;
    
    for(int i = 0; i < count; i+=32)
    {
        //Share 32 seeds with the block
        if(threadIdx.y == 0) 
        {
            shared_seeds[threadIdx.x] = seeds[i+threadIdx.x];
        }
        __syncthreads();
    	#pragma unroll 32
    	for(int j = 0; j < 32; j++)
        {
            //Use global index
    	    xdistance = shared_seeds[j].x -xglobal;
    	    ydistance = shared_seeds[j].y -yglobal;
    	    distance = (xdistance*xdistance)+(ydistance*ydistance);
            //If new distance < current distance, update colour
    	    if(distance < currentdistance)
            {
            	currentdistance = distance;
            	tempColor[0] = shared_seeds[j].color[0];
                tempColor[1] = shared_seeds[j].color[1];
                tempColor[2] = shared_seeds[j].color[2];
    	    }
            __syncthreads();
    	}
    }
    //Update global colour
    image[index] = tempColor[0];
    image[index+1] = tempColor[1];
    image[index+2] = tempColor[2];
}




static void Usage()
{
    printf("Usage: main [options]\n");
    printf("\t-n <num points>\n");
    printf("\t-i <inputfile>\t\tA list of 2-tuples (float, float) representing 2-d coordinates\n");
    printf("\t-o <outputfile.png>\n");
    printf("\t-w <width>\n");
    printf("\t-h <height>\n");
    printf("\t-p <pinned>\n");
}

int main(int argc, const char** argv) 
{
  //CUDA DECLARACIONES  
  unsigned int xnBlocks, ynBlocks, nThreads;
  
  //para calcular el tiempo en cuda
  float TiempoTotal, TiempoKernel;
  int cont_gpu;   //como maximo tenemos 4 devices por lo que es el maximo que podemos aprovechar
  hipEvent_t E0, E1, E2, E3;
  hipEvent_t X1, X2, X3;

  
  hipEventCreate(&E0);
  hipEventCreate(&E1);
  hipEventCreate(&E2);
  hipEventCreate(&E3);
  
    //Default values
    
    //Number of seeds
    int count = 64;
    
    // Image dimension
    int width = 512;
    int height = 512;
    int pinned = 1;
 
    //Files
    const char* inputfile = 0;
    const char* outputfile = "multigpu_rows.png";
    
    /*Change default values
        -i inputfyle
        -o outputfyle
        -n count
        -w width
        -h height
        -p pinned
    */  
    for( int i = 1; i < argc; ++i )
    {
        if(strcmp(argv[i], "-i") == 0)
        {
            if( i+1 < argc ) inputfile = argv[i+1];
            else
            {
                Usage();
                return 1;
            }
        }
        else if(strcmp(argv[i], "-o") == 0)
        {
            if( i+1 < argc ) outputfile = argv[i+1];
            else
            {
                Usage();
                return 1;
            }
        }
        else if(strcmp(argv[i], "-n") == 0)
        {
            if( i+1 < argc ) count = (int)atol(argv[i+1]);
            else
            {
                Usage();
                return 1;
            }
        }
        else if(strcmp(argv[i], "-w") == 0)
        {
            if( i+1 < argc ) width = (int)atol(argv[i+1]);
            else
            {
                Usage();
                return 1;
            }
        }
        else if(strcmp(argv[i], "-h") == 0)
        {
            if( i+1 < argc ) height = (int)atol(argv[i+1]);
            else
            {
                Usage();
                return 1;
            }
        }
        else if(strcmp(argv[i], "-p") == 0)
        {
            if( i+1 < argc ) pinned = (int)atol(argv[i+1]); // si es 1 es pinned, si es 0 no.
            else
            {
                Usage();
                return 1;
            }
        }
    }



    //Open file
    FILE* file = 0;
    if( inputfile )
    {
        if( strcmp(inputfile, "-") == 0 )
            file = stdin;
        file = fopen(inputfile, "r");
        if( !file )
        {
            fprintf(stderr, "Failed to open %s for reading\n", inputfile);
            return 1;
        }
        if(fscanf(file, "%d", &count)) 
        {
            printf("Points = %d\n", count);
        }
        else printf("Failed to read number of seeds\n");
    }

    printf("MULTIGPU_ROWS with %d seeds, %d pixels of width and %d pixels of height \n", count, width, height);


    seed* seeds;
    size_t seedssize = (size_t)(sizeof(seed) * count);
    if(pinned) hipHostMalloc((seed**)&seeds, seedssize);
    else seeds = (seed*)malloc(seedssize);
  
    
    //Get points from file
    if(inputfile)
    {
        for (int i = 0; i < count; ++i) 
        {
            if(fscanf(file,"%d %d", &seeds[i].x, &seeds[i].y));
        }
    }
    //If no input file, generate points
    else
    {
        int pointoffset = 10; // move the points inwards, for aestetic reasons

        srand(0);

        for( int i = 0; i < count; ++i )
        {
            seeds[i].x = (float)(pointoffset + rand() % (width-2*pointoffset));
            seeds[i].y = (float)(pointoffset + rand() % (height-2*pointoffset));
        }
    }


    //Generate colors for each seed
    for(int i = 0; i < count; ++i)
    {
        unsigned char basecolor = 120;
        seeds[i].color[0] = basecolor + (unsigned char)(rand() % (235 - basecolor));
        seeds[i].color[1] = basecolor + (unsigned char)(rand() % (235 - basecolor));
        seeds[i].color[2] = basecolor + (unsigned char)(rand() % (235 - basecolor));
    }



    //Initialize image
    size_t full_imagesize = (size_t)(width*height*3); //3 is for the RGB color of each pixel
    size_t imagesize = (size_t)(width*height*3)/4; //3 is for the RGB color of each pixel
    unsigned char* image;
    unsigned char* image0;
    unsigned char* image1;
    unsigned char* image2;
    unsigned char* image3;
    if(pinned) {
      hipHostMalloc((unsigned char**)&image,full_imagesize);
      hipHostMalloc((unsigned char**)&image0,imagesize);
      hipHostMalloc((unsigned char**)&image1,imagesize);
      hipHostMalloc((unsigned char**)&image2,imagesize);
      hipHostMalloc((unsigned char**)&image3,imagesize);
    }
    else {
      image = (unsigned char*)malloc(full_imagesize);
      image0 = (unsigned char*)malloc(imagesize);
      image1 = (unsigned char*)malloc(imagesize);
      image2 = (unsigned char*)malloc(imagesize);
      image3 = (unsigned char*)malloc(imagesize);
    }
    memset(image, 0, full_imagesize);
    memset(image0, 0, imagesize);
    memset(image1, 0, imagesize);
    memset(image2, 0, imagesize);
    memset(image3, 0, imagesize);

    hipGetDeviceCount(&cont_gpu);
    if (count < 4) { printf("No hay suficientes GPUs\n"); exit(0); }
    
    seed* device_seeds0;
    seed* device_seeds1;
    seed* device_seeds2;
    seed* device_seeds3;
    
    unsigned char* device_image0;
    unsigned char* device_image1;
    unsigned char* device_image2;
    unsigned char* device_image3;

    hipSetDevice(0);
    hipMalloc((unsigned char**)&device_image0, imagesize); 
    hipMalloc((seed**)&device_seeds0, seedssize); 

    
    hipSetDevice(1);
    hipMalloc((unsigned char**)&device_image1, imagesize); 
    hipMalloc((seed**)&device_seeds1, seedssize); 
    hipEventCreate(&X1); 
    
    hipSetDevice(2);
    hipMalloc((unsigned char**)&device_image2, imagesize); 
    hipMalloc((seed**)&device_seeds2, seedssize); 
    hipEventCreate(&X2); 
    
    hipSetDevice(3);
    hipMalloc((unsigned char**)&device_image3, imagesize); 
    hipMalloc((seed**)&device_seeds3, seedssize);
    hipEventCreate(&X3); 

    //lo de los tiempos lo hace el device0

    hipSetDevice(0);
    hipEventRecord(E0, 0);
   
    int height4 = height/4;
    nThreads = 32; 
    xnBlocks = width/nThreads;
    ynBlocks = height4/nThreads;

    dim3 dimGrid(xnBlocks, ynBlocks, 1);
    dim3 dimBlock(nThreads, nThreads, 1); 


    //Fill the image  
    //GPU 0                
    hipSetDevice(0);

    hipMemcpyAsync(device_seeds0, seeds, seedssize, hipMemcpyHostToDevice); 
    hipMemcpyAsync(device_image0, image0, imagesize, hipMemcpyHostToDevice);

    hipEventRecord(E1, 0);
    voronoi_cuda<<<dimGrid, dimBlock>>>(count, device_seeds0, width, height4, device_image0, 0);
    hipEventRecord(E2, 0); hipEventSynchronize(E2);

    hipMemcpyAsync(image, device_image0, imagesize, hipMemcpyDeviceToHost);

    //GPU 1
    hipSetDevice(1);

    hipMemcpyAsync(device_seeds1, seeds, seedssize, hipMemcpyHostToDevice); 
    hipMemcpyAsync(device_image1, image1, imagesize, hipMemcpyHostToDevice);

    voronoi_cuda<<<dimGrid, dimBlock>>>(count, device_seeds1, width, height4, device_image1, 1);

    hipMemcpyAsync(&image[imagesize], device_image1, imagesize, hipMemcpyDeviceToHost); 
    hipEventRecord(X1, 0);
    
    //GPU 2
    hipSetDevice(2);

    hipMemcpyAsync(device_seeds2, seeds, seedssize, hipMemcpyHostToDevice); 
    hipMemcpyAsync(device_image2, image2, imagesize, hipMemcpyHostToDevice);

    voronoi_cuda<<<dimGrid, dimBlock>>>(count, device_seeds2, width, height4, device_image2, 2);

    hipMemcpyAsync(&image[2*imagesize], device_image2, imagesize, hipMemcpyDeviceToHost); 
    hipEventRecord(X2, 0);


    //GPU 3
    hipSetDevice(3);

    hipMemcpyAsync(device_seeds3, seeds, seedssize, hipMemcpyHostToDevice); 
    hipMemcpyAsync(device_image3, image3, imagesize, hipMemcpyHostToDevice);

    voronoi_cuda<<<dimGrid, dimBlock>>>(count, device_seeds3, width, height4, device_image3, 3);

    hipMemcpyAsync(&image[3*imagesize], device_image3, imagesize, hipMemcpyDeviceToHost); 
    hipEventRecord(X3, 0);


    hipSetDevice(0);
    hipEventSynchronize(X1);
    hipEventSynchronize(X2);
    hipEventSynchronize(X3);
    
    hipEventRecord(E3, 0);
    hipEventSynchronize(E3);

    hipSetDevice(0);
    hipFree(device_seeds0);
    hipFree(device_image0);
    hipSetDevice(1);
    hipFree(device_seeds1);
    hipFree(device_image1);
    hipSetDevice(2);
    hipFree(device_seeds2);
    hipFree(device_image2);
    hipSetDevice(3);
    hipFree(device_seeds3);
    hipFree(device_image3);
        
    hipEventElapsedTime(&TiempoTotal,  E0, E3);
    hipEventElapsedTime(&TiempoKernel, E1, E2);
    
    printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
    printf("Tiempo Kernel: %4.6f milseg\n", TiempoKernel);
    hipSetDevice(0); hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);
    hipSetDevice(1); hipEventDestroy(X1);
    hipSetDevice(2); hipEventDestroy(X2);
    hipSetDevice(3); hipEventDestroy(X3);

    

    //Paint the seeds
    unsigned char color_seed[] = {255, 255, 255};
    int index;
    for(int i = 0; i < count; i++)
    {
        index = seeds[i].y*width*3+seeds[i].x*3;
        image[index] = color_seed[0];
        image[index+1] = color_seed[1];
        image[index+2] = color_seed[2];
    }


    //Transform tu png
    char path[512];
    sprintf(path, "%s", outputfile);
    stbi_write_png(path, width, height, 3, image, width*3);
    printf("Wrote image in %s\n", path);

    if (pinned) {
        hipHostFree(image); hipHostFree(image0); hipHostFree(image1); hipHostFree(image2); hipHostFree(image3);
	    hipHostFree(seeds);
    }
    else {
        free(image); free(image0); free(image1); free(image2); free(image3);
	    free(seeds);
    }
        
}
